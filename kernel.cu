#include "hip/hip_runtime.h"
#include "hipArray.h"

__global__ void kernel(float *x, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
		x[i] = sqrt(pow(3.14159,i));
	}
}

void hipArray::process()
{
	for(int i=0;i<8;++i)
	{
		hipMemcpyAsync(d_data[i],data[i],Size,hipMemcpyHostToDevice,cS[i]);
		kernel<<<1, 32, 0, cS[i]>>>(d_data[i], Len);
		hipMemcpyAsync(h_data[i],d_data[i],Size,hipMemcpyDeviceToHost,cS[i]);
	}
	hipDeviceSynchronize();

}
